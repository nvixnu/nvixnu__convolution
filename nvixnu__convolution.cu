#include "hip/hip_runtime.h"
#include "nvixnu__convolution.h"

void nvixnu__1d_convolution_host(double *input, double *output, const int length, const double *mask, const int mask_width){
	int ghosts_by_side = mask_width/2;
	double sum;
	int input_idx;

	for(int out_idx = 0; out_idx < length; out_idx++){ // Iterates through each output position to calculate it
		sum = 0;
		for(int mask_idx = 0; mask_idx < mask_width; mask_idx++){ // Iterates through each mask position
			input_idx = out_idx - ghosts_by_side + mask_idx; // Calculates the input index
			if(input_idx >= 0 && input_idx < length){ // Check if the input index is not a ghost
				sum+=input[input_idx]*mask[mask_idx]; //Performs the convolution
			}
		}
		output[out_idx] = sum;
	}
}

__global__
void nvixnu__1d_convolution_kernel(double *input, double *output, const int length, double *mask, const int mask_width){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	extern __shared__ double shared[];

	// Each thread loads data from global to the block shared memory
	shared[threadIdx.x] = tid < length ? input[tid] : 0.0;
	__syncthreads();

	// Defines the data index that belongs to each tile
	int this_tile_start_point = blockIdx.x * blockDim.x;
	int next_tile_start_point = (blockIdx.x + 1) * blockDim.x;

	// Go back int(mask_width/2) positions in order to start from the block scope external cells (halos or ghosts placed before the this_tile_start_point position)
	int n_start_point = tid - (mask_width/2);
	double p = 0;

	for(int j = 0; j < mask_width; j++){

		int n_index = n_start_point + j;
		if(n_index >= 0 && n_index < length){ //Check if the n_index not refers to a ghost cell
			if(n_index >= this_tile_start_point && n_index < next_tile_start_point){ // If is an internal cell (true) or a halo cell (false)
				p += shared[threadIdx.x + j - mask_width/2]*mask[j];
			}else{
				p += input[n_index] * mask[j]; //Takes the N[value] from the cache (Luckily!) or from the global memory and performs the convolution
			}
		}
	}
	output[tid] = p;
}
